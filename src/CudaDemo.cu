#include "hip/hip_runtime.h"
#include "include/CudaDemo.h"

#define NUM_REPAETS 10

CudaDemo::CudaDemo(/* args */)
{
}

CudaDemo::~CudaDemo()
{
}

void CudaDemo::SetGPU(){
        int iDeviceCount = 0;
    hipError_t error = hipGetDeviceCount(&iDeviceCount);

    if(error != hipSuccess||iDeviceCount==0){
        printf("No CUDA campatable GPU found!\n");
        exit(-1);
    }else{
        printf("The count of GPU is %d.\n",iDeviceCount);
    }

    int iDev = 0;
    error = hipSetDevice(iDev);
    if(error != hipSuccess){
        printf("File to set GPU 0 for computing.\n");
        exit(-1);
    }else{
        printf("Set GPU 0 for computing.\n");
    }
}

__global__ void hello_world(void){
    const int b_id = blockIdx.x;
    const int t_id = threadIdx.x;
    const int id = t_id + b_id*blockDim.x;
    printf("GPU: Hello World! -- block %d and thread %d -- global id %d\n",b_id,t_id,id);
}
void CudaDemo::Hello(){
    printf("CPU: Hello World!\n");
 
	hello_world<<<2, 5>>>();
 
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
    		printf("CUDA Error: %s\n", hipGetErrorString(err));
	} 
	hipDeviceReset();
}


__device__ float add(float* a,float *b){//设备函数只能被核函数或其他设备函数调用
    return *a+*b;
}
__global__ void addFromGPU(float *A, float *B, float *C, const int N)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int id = tid + bid * blockDim.x; 

    if(id>N){
        return;
    }

    // C[id] = A[id] + B[id];
    C[id] = add(&A[id],&B[id]);
    
}
void initialData(float *addr, int elemCount)
{
    for (int i = 0; i < elemCount; i++)
    {
        addr[i] = (float)(rand() & 0xFF) / 10.f;
    }
    return;
}
hipError_t ErrorCheck(hipError_t error_code, const char* filename, int lineNumber)
{
    if(error_code != hipSuccess)
    {  
        printf("cuDA error:\ncode=%d, name=%s, description=%s\nfile=%s, line=%d\n",
            error_code,hipGetErrorName(error_code),
            hipGetErrorString(error_code),filename, lineNumber);
        return error_code;
    }
    return error_code;
}
void CudaDemo::MatrixSum1D(){
    // 1、设置GPU设备
    SetGPU();

    // 2、分配主机内存和设备内存，并初始化
    int iElemCount = 512;                               // 设置元素数量
    size_t stBytesCount = iElemCount * sizeof(float);   // 字节数
    
    // （1）分配主机内存，并初始化
    float *fpHost_A, *fpHost_B, *fpHost_C;
    fpHost_A = (float *)malloc(stBytesCount);
    fpHost_B = (float *)malloc(stBytesCount);
    fpHost_C = (float *)malloc(stBytesCount);
    if (fpHost_A != NULL && fpHost_B != NULL && fpHost_C != NULL)
    {
        memset(fpHost_A, 0, stBytesCount);  // 主机内存初始化为0
        memset(fpHost_B, 0, stBytesCount);
        memset(fpHost_C, 0, stBytesCount);
    }
    else
    {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }

    // （2）分配设备内存，并初始化
    float *fpDevice_A, *fpDevice_B, *fpDevice_C;
    ErrorCheck(hipMalloc((float**)&fpDevice_A, stBytesCount),__FILE__,__LINE__);
    ErrorCheck(hipMalloc((float**)&fpDevice_B, stBytesCount),__FILE__,__LINE__);
    ErrorCheck(hipMalloc((float**)&fpDevice_C, stBytesCount),__FILE__,__LINE__);
    if (fpDevice_A != NULL && fpDevice_B != NULL && fpDevice_C != NULL)
    {
        ErrorCheck(hipMemset(fpDevice_A, 0, stBytesCount),__FILE__,__LINE__);  // 设备内存初始化为0
        ErrorCheck(hipMemset(fpDevice_B, 0, stBytesCount),__FILE__,__LINE__);
        ErrorCheck(hipMemset(fpDevice_C, 0, stBytesCount),__FILE__,__LINE__);
    }
    else
    {
        printf("fail to allocate memory\n");
        free(fpHost_A);
        free(fpHost_B);
        free(fpHost_C);
        exit(-1);
    }

    // 3、初始化主机中数据
    srand(666); // 设置随机种子
    initialData(fpHost_A, iElemCount);
    initialData(fpHost_B, iElemCount);
    
    // 4、数据从主机复制到设备
    ErrorCheck(hipMemcpy(fpDevice_A, fpHost_A, stBytesCount, hipMemcpyHostToDevice),__FILE__,__LINE__); 
    ErrorCheck(hipMemcpy(fpDevice_B, fpHost_B, stBytesCount, hipMemcpyHostToDevice),__FILE__,__LINE__); 
    ErrorCheck(hipMemcpy(fpDevice_C, fpHost_C, stBytesCount, hipMemcpyHostToDevice),__FILE__,__LINE__);


    // 5、调用核函数在设备中进行计算
    dim3 block(32);
    dim3 grid(iElemCount / 32);

    float t_sum = 0;
    for(int repeat = 0;repeat<NUM_REPAETS;repeat++){
        hipEvent_t start,stop;
        ErrorCheck(hipEventCreate(&start),__FILE__,__LINE__);
        ErrorCheck(hipEventCreate(&stop),__FILE__,__LINE__);
        ErrorCheck(hipEventRecord(start),__FILE__,__LINE__);
        hipEventQuery(start);

        addFromGPU<<<grid, block>>>(fpDevice_A, fpDevice_B, fpDevice_C, iElemCount);    // 调用核函数
        //检查核函数错误
        ErrorCheck(hipGetLastError(),__FILE__,__LINE__);
        ErrorCheck(hipDeviceSynchronize(),__FILE__,__LINE__);

        ErrorCheck(hipEventRecord(stop),__FILE__,__LINE__);
        ErrorCheck(hipEventSynchronize(stop),__FILE__,__LINE__);
        float elapsed_time;
        ErrorCheck(hipEventElapsedTime(&elapsed_time,start,stop),__FILE__,__LINE__);

        if(repeat>0){
            t_sum+=elapsed_time;
        }
        ErrorCheck(hipEventDestroy(start),__FILE__,__LINE__);
        ErrorCheck(hipEventDestroy(stop),__FILE__,__LINE__);
    }
    const float t_ave = t_sum/NUM_REPAETS;
    printf("Time = %g ms.\n",t_ave);

    // 6、将计算得到的数据从设备传给主机
    ErrorCheck(hipMemcpy(fpHost_C, fpDevice_C, stBytesCount, hipMemcpyDeviceToHost),__FILE__,__LINE__);


    for (int i = 0; i < 10; i++)    // 打印
    {
        printf("idx=%2d\tmatrix_A:%.2f\tmatrix_B:%.2f\tresult=%.2f\n", i+1, fpHost_A[i], fpHost_B[i], fpHost_C[i]);
    }

    // 7、释放主机与设备内存
    free(fpHost_A);
    free(fpHost_B);
    free(fpHost_C);
    ErrorCheck(hipFree(fpDevice_A),__FILE__,__LINE__);
    ErrorCheck(hipFree(fpDevice_B),__FILE__,__LINE__);
    ErrorCheck(hipFree(fpDevice_C),__FILE__,__LINE__);

    hipDeviceReset();
}

void CudaDemo::GetGPUProperties(int device_id){
    ErrorCheck(hipSetDevice(device_id), __FILE__, __LINE__);

    hipDeviceProp_t prop;
    ErrorCheck(hipGetDeviceProperties(&prop, device_id), __FILE__, __LINE__);

    printf("Device id:                                 %d\n",
        device_id);
    printf("Device name:                               %s\n",
        prop.name);
    printf("Compute capability:                        %d.%d\n",
        prop.major, prop.minor);
    printf("Amount of global memory:                   %g GB\n",
        prop.totalGlobalMem / (1024.0 * 1024 * 1024));
    printf("Amount of constant memory:                 %g KB\n",
        prop.totalConstMem  / 1024.0);
    printf("Maximum grid size:                         %d %d %d\n",
        prop.maxGridSize[0], 
        prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Maximum block size:                        %d %d %d\n",
        prop.maxThreadsDim[0], prop.maxThreadsDim[1], 
        prop.maxThreadsDim[2]);
    printf("Number of SMs:                             %d\n",
        prop.multiProcessorCount);
    printf("Maximum amount of shared memory per block: %g KB\n",
        prop.sharedMemPerBlock / 1024.0);
    printf("Maximum amount of shared memory per SM:    %g KB\n",
        prop.sharedMemPerMultiprocessor / 1024.0);
    printf("Maximum number of registers per block:     %d K\n",
        prop.regsPerBlock / 1024);
    printf("Maximum number of registers per SM:        %d K\n",
        prop.regsPerMultiprocessor / 1024);
    printf("Maximum number of threads per block:       %d\n",
        prop.maxThreadsPerBlock);
    printf("Maximum number of threads per SM:          %d\n",
        prop.maxThreadsPerMultiProcessor);
}

int CudaDemo::GetSPcores(int device_id){
    ErrorCheck(hipSetDevice(device_id), __FILE__, __LINE__);
    hipDeviceProp_t devProp;
    ErrorCheck(hipGetDeviceProperties(&devProp, device_id), __FILE__, __LINE__);

    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
     case 2: // Fermi
      if (devProp.minor == 1) cores = mp * 48;
      else cores = mp * 32;
      break;
     case 3: // Kepler
      cores = mp * 192;
      break;
     case 5: // Maxwell
      cores = mp * 128;
      break;
     case 6: // Pascal
      if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
      else if (devProp.minor == 0) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 7: // Volta and Turing
      if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 8: // Ampere
      if (devProp.minor == 0) cores = mp * 64;
      else if (devProp.minor == 6) cores = mp * 128;
      else if (devProp.minor == 9) cores = mp * 128; // ada lovelace
      else printf("Unknown device type\n");
      break;
     case 9: // Hopper
      if (devProp.minor == 0) cores = mp * 128;
      else printf("Unknown device type\n");
      break;
     default:
      printf("Unknown device type\n"); 
      break;
      }
    printf("cores num:%d.\n",cores);
    return cores;
}

__global__ void addMatrix(int *A, int *B, int *C, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < nx)
    {
        for (int iy = 0; iy < ny; iy++)
        {
            int idx = iy * nx + ix;
            C[idx] = A[idx] + B[idx];
        }
        
    }
}
void CudaDemo::Grid1d_bLOCK1D(){
    // 1、设置GPU设备
    SetGPU();

    // 2、分配主机内存和设备内存，并初始化
    int nx = 16;
    int ny = 8;
    int nxy = nx * ny;
    size_t stBytesCount = nxy * sizeof(int);
     
     // （1）分配主机内存，并初始化
    int *ipHost_A, *ipHost_B, *ipHost_C;
    ipHost_A = (int *)malloc(stBytesCount);
    ipHost_B = (int *)malloc(stBytesCount);
    ipHost_C = (int *)malloc(stBytesCount);
    if (ipHost_A != NULL && ipHost_B != NULL && ipHost_C != NULL)
    {
        for (int i = 0; i < nxy; i++)
            {
                ipHost_A[i] = i;
                ipHost_B[i] = i + 1;
            }
        memset(ipHost_C, 0, stBytesCount); 
    }
    else
    {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }
    

    // （2）分配设备内存，并初始化
    int *ipDevice_A, *ipDevice_B, *ipDevice_C;
    ErrorCheck(hipMalloc((int**)&ipDevice_A, stBytesCount), __FILE__, __LINE__); 
    ErrorCheck(hipMalloc((int**)&ipDevice_B, stBytesCount), __FILE__, __LINE__); 
    ErrorCheck(hipMalloc((int**)&ipDevice_C, stBytesCount), __FILE__, __LINE__); 
    if (ipDevice_A != NULL && ipDevice_B != NULL && ipDevice_C != NULL)
    {
        ErrorCheck(hipMemcpy(ipDevice_A, ipHost_A, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
        ErrorCheck(hipMemcpy(ipDevice_B, ipHost_B, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
        ErrorCheck(hipMemcpy(ipDevice_C, ipHost_C, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
    }   
    else
    {
        printf("Fail to allocate memory\n");
        free(ipHost_A);
        free(ipHost_B);
        free(ipHost_C);
        exit(1);
    }

    // calculate on GPU
    dim3 block(4, 1);
    dim3 grid((nx + block.x -1) / block.x, 1);
    printf("Thread config:grid:<%d, %d>, block:<%d, %d>\n", grid.x, grid.y, block.x, block.y);
    
    addMatrix<<<grid, block>>>(ipDevice_A, ipDevice_B, ipDevice_C, nx, ny);  // 调用内核函数
    
    ErrorCheck(hipMemcpy(ipHost_C, ipDevice_C, stBytesCount, hipMemcpyDeviceToHost), __FILE__, __LINE__); 
    for (int i = 0; i < 10; i++)
    {
        printf("id=%d, matrix_A=%d, matrix_B=%d, result=%d\n", i + 1,ipHost_A[i], ipHost_B[i], ipHost_C[i]);
    }

    free(ipHost_A);
    free(ipHost_B);
    free(ipHost_C);

    ErrorCheck(hipFree(ipDevice_A), __FILE__, __LINE__); 
    ErrorCheck(hipFree(ipDevice_B), __FILE__, __LINE__); 
    ErrorCheck(hipFree(ipDevice_C), __FILE__, __LINE__); 

    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__); 
}
__global__ void addMatrix2(int *A, int *B, int *C, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = blockIdx.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        C[idx] = A[idx] + B[idx];
    }
}
void CudaDemo::Grid2d_bLOCK1D(){
    // 1、设置GPU设备
    SetGPU();

    // 2、分配主机内存和设备内存，并初始化
    int nx = 16;
    int ny = 8;
    int nxy = nx * ny;
    size_t stBytesCount = nxy * sizeof(int);
     
     // （1）分配主机内存，并初始化
    int *ipHost_A, *ipHost_B, *ipHost_C;
    ipHost_A = (int *)malloc(stBytesCount);
    ipHost_B = (int *)malloc(stBytesCount);
    ipHost_C = (int *)malloc(stBytesCount);
    if (ipHost_A != NULL && ipHost_B != NULL && ipHost_C != NULL)
    {
        for (int i = 0; i < nxy; i++)
            {
                ipHost_A[i] = i;
                ipHost_B[i] = i + 1;
            }
        memset(ipHost_C, 0, stBytesCount); 
    }
    else
    {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }
    

    // （2）分配设备内存，并初始化
    int *ipDevice_A, *ipDevice_B, *ipDevice_C;
    ErrorCheck(hipMalloc((int**)&ipDevice_A, stBytesCount), __FILE__, __LINE__); 
    ErrorCheck(hipMalloc((int**)&ipDevice_B, stBytesCount), __FILE__, __LINE__); 
    ErrorCheck(hipMalloc((int**)&ipDevice_C, stBytesCount), __FILE__, __LINE__); 
    if (ipDevice_A != NULL && ipDevice_B != NULL && ipDevice_C != NULL)
    {
        ErrorCheck(hipMemcpy(ipDevice_A, ipHost_A, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
        ErrorCheck(hipMemcpy(ipDevice_B, ipHost_B, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
        ErrorCheck(hipMemcpy(ipDevice_C, ipHost_C, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
    }   
    else
    {
        printf("Fail to allocate memory\n");
        free(ipHost_A);
        free(ipHost_B);
        free(ipHost_C);
        exit(1);
    }

    // calculate on GPU
    dim3 block(4, 1);
    dim3 grid((nx + block.x -1) / block.x, ny);
    printf("Thread config:grid:<%d, %d>, block:<%d, %d>\n", grid.x, grid.y, block.x, block.y);
    
    addMatrix2<<<grid, block>>>(ipDevice_A, ipDevice_B, ipDevice_C, nx, ny);  // 调用内核函数
    
    ErrorCheck(hipMemcpy(ipHost_C, ipDevice_C, stBytesCount, hipMemcpyDeviceToHost), __FILE__, __LINE__); 
    for (int i = 0; i < 10; i++)
    {
        printf("id=%d, matrix_A=%d, matrix_B=%d, result=%d\n", i + 1,ipHost_A[i], ipHost_B[i], ipHost_C[i]);
    }

    free(ipHost_A);
    free(ipHost_B);
    free(ipHost_C);

    ErrorCheck(hipFree(ipDevice_A), __FILE__, __LINE__); 
    ErrorCheck(hipFree(ipDevice_B), __FILE__, __LINE__); 
    ErrorCheck(hipFree(ipDevice_C), __FILE__, __LINE__); 

    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__); 
}
__global__ void addMatrix3(int *A, int *B, int *C, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        C[idx] = A[idx] + B[idx];
    }
}
void CudaDemo::Grid2d_bLOCK2D(){
    // 1、设置GPU设备
    SetGPU();

    // 2、分配主机内存和设备内存，并初始化
    int nx = 16;
    int ny = 8;
    int nxy = nx * ny;
    size_t stBytesCount = nxy * sizeof(int);
     
     // （1）分配主机内存，并初始化
    int *ipHost_A, *ipHost_B, *ipHost_C;
    ipHost_A = (int *)malloc(stBytesCount);
    ipHost_B = (int *)malloc(stBytesCount);
    ipHost_C = (int *)malloc(stBytesCount);
    if (ipHost_A != NULL && ipHost_B != NULL && ipHost_C != NULL)
    {
        for (int i = 0; i < nxy; i++)
            {
                ipHost_A[i] = i;
                ipHost_B[i] = i + 1;
            }
        memset(ipHost_C, 0, stBytesCount); 
    }
    else
    {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }
    

    // （2）分配设备内存，并初始化
    int *ipDevice_A, *ipDevice_B, *ipDevice_C;
    ErrorCheck(hipMalloc((int**)&ipDevice_A, stBytesCount), __FILE__, __LINE__); 
    ErrorCheck(hipMalloc((int**)&ipDevice_B, stBytesCount), __FILE__, __LINE__); 
    ErrorCheck(hipMalloc((int**)&ipDevice_C, stBytesCount), __FILE__, __LINE__); 
    if (ipDevice_A != NULL && ipDevice_B != NULL && ipDevice_C != NULL)
    {
        ErrorCheck(hipMemcpy(ipDevice_A, ipHost_A, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
        ErrorCheck(hipMemcpy(ipDevice_B, ipHost_B, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
        ErrorCheck(hipMemcpy(ipDevice_C, ipHost_C, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
    }   
    else
    {
        printf("Fail to allocate memory\n");
        free(ipHost_A);
        free(ipHost_B);
        free(ipHost_C);
        exit(1);
    }

    // calculate on GPU
    dim3 block(4, 4);
    dim3 grid((nx + block.x -1) / block.x, (ny + block.y - 1) / block.y);
    printf("Thread config:grid:<%d, %d>, block:<%d, %d>\n", grid.x, grid.y, block.x, block.y);
    
    addMatrix3<<<grid, block>>>(ipDevice_A, ipDevice_B, ipDevice_C, nx, ny);  // 调用内核函数
    
    ErrorCheck(hipMemcpy(ipHost_C, ipDevice_C, stBytesCount, hipMemcpyDeviceToHost), __FILE__, __LINE__); 
    for (int i = 0; i < 10; i++)
    {
        printf("id=%d, matrix_A=%d, matrix_B=%d, result=%d\n", i + 1,ipHost_A[i], ipHost_B[i], ipHost_C[i]);
    }

    free(ipHost_A);
    free(ipHost_B);
    free(ipHost_C);

    ErrorCheck(hipFree(ipDevice_A), __FILE__, __LINE__); 
    ErrorCheck(hipFree(ipDevice_B), __FILE__, __LINE__); 
    ErrorCheck(hipFree(ipDevice_C), __FILE__, __LINE__); 

    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__); 
}